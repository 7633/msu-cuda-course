
#include <hip/hip_runtime.h>
#define	N	(1024*1024)		// array size
#define	PI	3.1415926f

__global__ void kernel ( float * data )
{ 
   int 		idx = blockIdx.x * blockDim.x + threadIdx.x;
   float	x   = 2.0f * PI * (float) idx / (float) N;
   
   data [idx] = sinf ( sqrtf ( x ) );
}

int main ( int argc, char *  argv [] )
{
    float * a   = new float [N];	// CPU memory
    float * dev = NULL;				// GPU memory
		
									// allocate device memory
    hipMalloc ( (void**)&dev, N * sizeof ( float ) );

    dim3 threads = dim3( 512, 1 );
    dim3 blocks  = dim3( N / threads.x, 1 );
					
    kernel<<<blocks, threads>>> ( dev );

    hipMemcpy ( a, dev, N * sizeof ( float ), hipMemcpyDeviceToHost );
    hipFree   ( dev   );

    delete a;

    return 0;
}
