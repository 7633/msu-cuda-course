#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define	PI	3.1415926f

#define SQR(x) ((x)*(x))

#define CUDA_SAFE_CALL(call)                                          \
    {                                                                 \
        cudaResult = call;                                            \
        if (cudaResult != hipSuccess)                                \
        fprintf(stderr, "cuda error at line %d\n", __LINE__);         \
    }                                                                 \

#define CUDA_CHECK_CALL_SYNC(call)                                    \
    {                                                                 \
        call;                                                         \
        hipDeviceSynchronize();                                      \
        cudaResult = hipGetLastError();                              \
        if (cudaResult != hipSuccess)                                \
        fprintf(stderr, "cuda error at line %d\n", __LINE__);         \
    }                                                                 \

#define CUDA_CHECK_CALL_ASYNC(call)                                   \
    {                                                                 \
        call;                                                         \
        cudaResult = hipGetLastError();                              \
        if (cudaResult != hipSuccess)                                \
        fprintf(stderr, "cuda error at line %d\n", __LINE__);         \
    }                                                                 \

texture<float, 2, hipReadModeElementType> g_TexRef;

__global__ void kernel ( float * data )
{ 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    data [idx + blockIdx.y * gridDim.x * blockDim.x] = tex2D(g_TexRef, idx, blockIdx.y);
}

int main ( int argc, char *  argv [] )
{
    hipError_t cudaResult = hipSuccess;

    int nThreads  = 64;
    int nBlocksX  = 256;
    int nBlocksY  = 2;
    int nElem    = nThreads * nBlocksX * nBlocksY;
    int nMemSizeInBytes = nElem * sizeof(float);

    float * phA = NULL;             // host    memory    pointer
    float * phB = NULL;             // host    memory    pointer
    float * pdA = NULL;				// device  memory    pointer
    float * pdB = NULL;				// device  memory    pointer
    hipArray * paA = NULL;         // device  hipArray pointer

    hipChannelFormatDesc cfDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    phA = (float *) malloc(nMemSizeInBytes);
    phB = (float *) malloc(nMemSizeInBytes);
    CUDA_SAFE_CALL( hipMalloc ( (void**) &pdA, nMemSizeInBytes ) );
    CUDA_SAFE_CALL( hipMalloc ( (void**) &pdB, nMemSizeInBytes ) );
    CUDA_SAFE_CALL( hipMallocArray(&paA, &cfDesc, nBlocksX * nThreads, nBlocksY) );

    for (int idx = 0; idx < nThreads * nBlocksX; idx++)
        phA[idx] = sinf(idx * 2.0f * PI / (nThreads * nBlocksX) );
    
    for (int idx = 0; idx < nThreads * nBlocksX; idx++)
        phA[idx + nThreads * nBlocksX] = cosf(idx * 2.0f * PI / (nThreads * nBlocksX) );

    CUDA_SAFE_CALL( hipMemcpyToArray ( paA, 0, 0, phA, nMemSizeInBytes, hipMemcpyHostToDevice ) );

    CUDA_SAFE_CALL( hipBindTextureToArray(g_TexRef, paA) );

    dim3 threads = dim3( nThreads );
    dim3 blocks  = dim3( nBlocksX, nBlocksY );

    kernel<<<blocks, threads>>> ( pdB );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUDA_SAFE_CALL( hipMemcpy ( phB, pdB, nMemSizeInBytes, hipMemcpyDeviceToHost ) );

    for (int idx = 0; idx < nThreads * nBlocksX * nBlocksY; idx++)
        if (SQR(phA[idx] - phB[idx]) > 0.0001f) printf("a[%d] = %.5f != %.5f = b[%d]\n", idx, phA[idx], phB[idx], idx);

    free(phA);
    free(phB);

    CUDA_SAFE_CALL( hipFree ( pdA ) );
    CUDA_SAFE_CALL( hipFree ( pdB ) );
    CUDA_SAFE_CALL( hipFreeArray( paA ) );

    return 0;
}
