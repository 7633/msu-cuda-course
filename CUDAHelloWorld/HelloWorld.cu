#include "hip/hip_runtime.h"
#include "HelloWorld.h"

__global__ 
void CU_SimpleAddKernel( float * pA, float * pB, float * pC)
{
    int tid = (threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z));

    int bid = (blockIdx.x  + gridDim.x  * (blockIdx.y  + gridDim.z  * blockIdx.z ));
    int blockSize = blockDim.x * blockDim.y * blockDim.z;

    int index = bid * blockSize + tid;
    
    pC[index] = pA[index] + pB[index];
}


float CU_SimpleAddKernel( float * pA, float * pB, float * pC, int * pthreads, int * pblocks)
{
    dim3 threads = dim3(pthreads[0], pthreads[1], pthreads[2]);
    dim3 blocks  = dim3(pblocks[0], pblocks[1]);

	// create cuda event handles
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate ( &start );
    hipEventCreate ( &stop );
	
	// asynchronously issue work to the GPU (all to stream 0)
    hipEventRecord ( start, 0 );
	
    CU_SimpleAddKernel<<<blocks, threads>>>(pA, pB, pC);
	
    hipEventRecord ( stop, 0 );

	hipEventSynchronize ( stop );
	hipEventElapsedTime ( &gpuTime, start, stop );

    hipEventDestroy ( start );
    hipEventDestroy ( stop  );

    return gpuTime;
}
