
#include <hip/hip_runtime.h>
#include <stdio.h>

int main ( int argc, char *  argv [] )
{
	int				deviceCount;
	hipDeviceProp_t	devProp;
	
	hipGetDeviceCount ( &deviceCount );
	
	printf ( "Found %d devices\n", deviceCount );
	
	for ( int device = 0; device < deviceCount; device++ )
	{
		hipGetDeviceProperties ( &devProp, device );
		
		printf ( "Device %d\n", device );
		printf ( "Compute capability     : %d.%d\n", devProp.major, devProp.minor );
		printf ( "Name                   : %s\n", devProp.name );
		printf ( "Total Global Memory    : %d\n", devProp.totalGlobalMem );
		printf ( "Shared memory per block: %d\n", devProp.sharedMemPerBlock );
		printf ( "Registers per block    : %d\n", devProp.regsPerBlock );
		printf ( "Warp size              : %d\n", devProp.warpSize );
		printf ( "Max threads per block  : %d\n", devProp.maxThreadsPerBlock );
		printf ( "Total constant memory  : %d\n", devProp.totalConstMem );
	}
	
    return 0;
}
