#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define	PI	3.1415926f

#define SQR(x) ((x)*(x))

#define CUDA_SAFE_CALL(call)                                          \
    {                                                                 \
        cudaResult = call;                                            \
        if (cudaResult != hipSuccess)                                \
        fprintf(stderr, "cuda error at line %d\n", __LINE__);         \
    }                                                                 \

#define CUDA_CHECK_CALL_SYNC(call)                                    \
    {                                                                 \
        call;                                                         \
        hipDeviceSynchronize();                                      \
        cudaResult = hipGetLastError();                              \
        if (cudaResult != hipSuccess)                                \
        fprintf(stderr, "cuda error at line %d\n", __LINE__);         \
    }                                                                 \

#define CUDA_CHECK_CALL_ASYNC(call)                                   \
    {                                                                 \
        call;                                                         \
        cudaResult = hipGetLastError();                              \
        if (cudaResult != hipSuccess)                                \
        fprintf(stderr, "cuda error at line %d\n", __LINE__);         \
    }                                                                 \

texture<float, 1, hipReadModeElementType> g_TexRef;

__global__ void kernel ( float * data )
{ 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    data [idx] = tex1Dfetch(g_TexRef, idx);
}

int main ( int argc, char *  argv [] )
{
    hipError_t cudaResult = hipSuccess;

    int nThreads = 64;
    int nBlocks  = 256;
    int nElem    = nThreads * nBlocks;
    int nMemSizeInBytes = nElem * sizeof(float);

    float * phA = NULL;             // host    memory  pointer
    float * phB = NULL;             // host    memory  pointer
    float * pdA = NULL;				// device  memory  pointer
    float * pdB = NULL;				// device  memory  pointer

    phA = (float *) malloc(nMemSizeInBytes);
    phB = (float *) malloc(nMemSizeInBytes);
    CUDA_SAFE_CALL( hipMalloc ( (void**) &pdA, nMemSizeInBytes ) );
    CUDA_SAFE_CALL( hipMalloc ( (void**) &pdB, nMemSizeInBytes ) );

    for (int idx = 0; idx < nThreads * nBlocks; idx++)
        phA[idx] = sinf(idx * 2.0f * PI / (nThreads * nBlocks) );

    CUDA_SAFE_CALL( hipMemcpy ( pdA, phA, nMemSizeInBytes, hipMemcpyHostToDevice ) );

    CUDA_SAFE_CALL( hipBindTexture(0, g_TexRef, pdA, nMemSizeInBytes) );

    dim3 threads = dim3( nThreads );
    dim3 blocks  = dim3( nBlocks );

    kernel<<<blocks, threads>>> ( pdB );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUDA_SAFE_CALL( hipMemcpy ( phB, pdB, nMemSizeInBytes, hipMemcpyDeviceToHost ) );

    for (int idx = 0; idx < nThreads * nBlocks; idx++)
        if (SQR(phA[idx] - phB[idx]) > 0.0001f) printf("a[%d] = %.5f != %.5f = b[%d]\n", idx, phA[idx], phB[idx], idx);

    free(phA);
    free(phB);

    CUDA_SAFE_CALL( hipFree ( pdA ) );
    CUDA_SAFE_CALL( hipFree ( pdB ) );

    return 0;
}
