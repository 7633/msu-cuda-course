#include "hip/hip_runtime.h"
#include <stdio.h>

#define	N	(512*512)		// array size
#define	PI	3.1415926f

__global__ void kernel ( float * data )
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   
   data [idx] = idx;
}

int main ( int argc, char *  argv [] )
{
    float * a   = new float [N];	// CPU memory
    float * dev = NULL;				// GPU memory
		
									// allocate device memory
    hipMalloc ( (void**)&dev, N * sizeof ( float ) );

    dim3 threads = dim3( 512, 1 );
    dim3 blocks  = dim3( N / threads.x, 1 );
					
    kernel<<<blocks, threads>>> ( dev );
    hipDeviceSynchronize();

    hipMemcpy ( a, dev, N * sizeof ( float ), hipMemcpyDeviceToHost );
    hipFree   ( dev   );

    //for (int idx = 0; idx < N; idx++)
    //    printf("a[%d] = %.5f\n", idx, a[idx]);

    delete [] a;

    return 0;
}
